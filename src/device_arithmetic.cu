#include "hip/hip_runtime.h"
#include <device_arithmetic.h>

#define dmat device_matrix
#define dvec thrust::device_vector

template <typename T>
dmat<T> operator * (const dvec<T>& col_vector, const dvec<T>& row_vector) {
  size_t m = col_vector.size();
  size_t n = row_vector.size();
  dmat<T> result(m, n);
  size_t k = 1;

  // Treat device_vector as an 1 by N matrix
  const T* cv = thrust::raw_pointer_cast(col_vector.data());
  const T* rv = thrust::raw_pointer_cast(row_vector.data());

  float alpha = 1.0, beta = 0.0;

  int lda = m;
  int ldb = 1;
  int ldc = m;

  hipblasStatus_t status;
  status = hipblasSgemm(CUBLAS_HANDLE::getInstance(), HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, cv, lda, rv, ldb, &beta, result.getData(), ldc);

  CCE(status);

  return result;
}

template <typename T>
dvec<T> operator & (const dvec<T>& x, const dvec<T>& y) {
  assert(x.size() == y.size());
  dvec<T> z(x.size());
  thrust::transform(x.begin(), x.end(), y.begin(), z.begin(), thrust::multiplies<T>());
  return z;
}


template <typename T>
dmat<T> operator * (const dvec<T>& v, const dmat<T>& A) {
  assert(v.size() == A.getRows());
  device_matrix<T> m(1, A.getCols());

  float alpha = 1.0, beta = 0.0;
  CCE(hipblasSgemv(CUBLAS_HANDLE::getInstance(), HIPBLAS_OP_T, A.getRows(), A.getCols(), &alpha, A.getData(), A.getRows(), thrust::raw_pointer_cast(v.data()), STRIDE, &beta, m.getData(), STRIDE));

  return m;
}

template <typename T>
dmat<T> operator * (const dmat<T>& A, const dvec<T>& v) {
  assert(A.getCols() == v.size());

  device_matrix<T> m(A.getRows(), 1);

  float alpha = 1.0, beta = 0.0;
  CCE(hipblasSgemv(CUBLAS_HANDLE::getInstance(), HIPBLAS_OP_N, A.getRows(), A.getCols(), &alpha, A.getData(), A.getRows(), thrust::raw_pointer_cast(v.data()), STRIDE, &beta, m.getData(), STRIDE));

  return m;
}

template <typename T>
dmat<T> operator + (T val, const dmat<T>& m) {
  return m + val;
}

template <typename T>
T norm(const thrust::host_vector<T>& v) {
  return std::sqrt( thrust::transform_reduce(v.begin(), v.end(), func::square<T>(), 0, thrust::plus<T>()) );
}

template <typename T>
T norm(const thrust::device_vector<T>& v) {
  return std::sqrt( thrust::transform_reduce(v.begin(), v.end(), func::square<T>(), 0, thrust::plus<T>()) );
}

template <typename T>
void print(const thrust::host_vector<T>& v) {
  std::vector<T> stl_v(v.begin(), v.end());
  printf("[");
  for (size_t i=0; i<v.size(); ++i)
    printf("%.4f ", v[i]);
  printf("]\n");
}

template <typename T>
void print(const thrust::device_vector<T>& v) {
  thrust::host_vector<T> hv(v);
  print(hv);
}

template <typename T>
dmat<T> operator & (const dvec<T>& v, const dmat<T>& m) {

  // !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  // FIXME !!!!!!!!!!!!!! THIS IS FUCKING SLOW !!!!!!!!!!!!!!!!! FIXME
  // !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  assert(v.size() == m.getCols());
  dmat<T> result(m);

  size_t rows = m.getRows();
  size_t cols = m.getCols();

  for (size_t i=0; i<cols; ++i) {
    thrust::device_ptr<T> ptr(m.getData() + rows * i);
    thrust::device_ptr<T> ptr2(result.getData() + rows * i);

    dvec<T> cv(ptr, ptr + rows);
    dvec<T> cv2(ptr2, ptr2 + rows);

    thrust::transform(
	m.getData() + rows * i,
	m.getData() + rows * (i + 1),
	result.getData() + rows * i,
	func::ax<T>(v[i]) );
  }

  return result;
}

#define EXPLICITLY_INSTANTIATE(T) \
template T norm<T> (const thrust::host_vector<T>& v); \
template T norm<T> (const thrust::device_vector<T>& v); \
template void print<T> (const thrust::host_vector<T>& v); \
template void print<T> (const thrust::device_vector<T>& v); \
template dmat<T> operator & <T> (const dvec<T>& v, const dmat<T>& m); \
template dmat<T> operator * <T> (const dvec<T>& col_vector, const dvec<T>& row_vector); \
template dvec<T> operator & <T> (const dvec<T>& x, const dvec<T>& y); \
template dmat<T> operator * <T> (const dvec<T>& v, const dmat<T>& m); \
template dmat<T> operator * <T> (const dmat<T>& m, const dvec<T>& v);

EXPLICITLY_INSTANTIATE(float);

#undef dmat
#undef dvec
