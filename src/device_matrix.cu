#include <device_matrix.h>
#define mylog(token) {cout << #token " = " << token << endl;}

// ===============================
// ===== class device_matrix =====
// ===============================

template <typename T>
device_matrix<T>::device_matrix(): _rows(0), _cols(0), _data(NULL) { }

template <typename T>
device_matrix<T>::device_matrix(size_t r, size_t c): _rows(r), _cols(c), _data(NULL) {
  _init();
  fillwith(0);
}

template <typename T>
device_matrix<T>::device_matrix(const string& filename): _rows(0), _cols(0), _data(NULL) {

  const size_t MAX_BUFFER = 65536;
  char line[MAX_BUFFER];

  FILE* fid = fopen(filename.c_str(), "r");
  while (fgets(line, MAX_BUFFER, fid)) {
    _rows++;

    if (_cols != 0)
      continue;

    char* token = strtok(line, " \n");
    ++_cols;
    while(strtok(NULL, " \n"))
      ++_cols;
  }
  fseek(fid, 0, SEEK_SET);

  // BEWARE !!
  // BLAS stores data in column-major
  T* data = new T[_rows*_cols];
  for (size_t i=0; i<_rows; ++i)
    for (size_t j=0; j<_cols; ++j)
      fscanf(fid, "%f ", &(data[j*_rows + i]));
  fclose(fid);

  _init();
  CCE(hipMemcpy(_data, data, sizeof(T) * _rows * _cols, hipMemcpyHostToDevice));
  delete [] data;
}
// Copy Constructor 
template <typename T>
device_matrix<T>::device_matrix(const device_matrix<T>& source): _rows(source._rows), _cols(source._cols), _data(NULL) {
  _init();
  CCE(hipMemcpy(_data, source._data, sizeof(T) * _rows * _cols, hipMemcpyDeviceToDevice));
}

#ifdef HAVE_THRUST_DEVICE_VECTOR_H
// Conversion operator
template <typename T>
device_matrix<T>::operator thrust::device_vector<T>() const {
  assert(_rows == 1 || _cols == 1);
  return thrust::device_vector<T>(_data, _data + size());
}
#endif

template <typename T>
device_matrix<T>::~device_matrix() {
  CCE(hipFree(_data));
}

// ===========================
// ===== Other Functions =====
// ===========================

// ===== Addition =====
template <typename T>
device_matrix<T>& device_matrix<T>::operator += (T val) {
  CCE(hipblasSaxpy(CUBLAS_HANDLE::getInstance(), _rows*_cols, &val, SCALAR_MEMORY_BUFFER<T>::getBuffer(), 0, _data, 1));
  return *this;
} 

template <typename T>
device_matrix<T> device_matrix<T>::operator + (T val) const {
  device_matrix<T> m(*this);
  return (m += val);
}

template <typename T>
device_matrix<T>& device_matrix<T>::operator += (const device_matrix<T>& rhs) {
  *this = *this + rhs;
  return *this;
}

template <typename T>
device_matrix<T> device_matrix<T>::operator + (const device_matrix<T>& rhs) const {
  device_matrix<T> result(_rows, _cols);
  sgeam(*this, rhs, result, 1.0, 1.0);
  return result;
}

// ===== Substraction =====
template <typename T>
device_matrix<T>& device_matrix<T>::operator -= (T val) {
  val = -val;
  CCE(hipblasSaxpy(CUBLAS_HANDLE::getInstance(), _rows*_cols, &val, SCALAR_MEMORY_BUFFER<T>::getBuffer(), 0, _data, 1));
  return *this;
}

template <typename T>
device_matrix<T> device_matrix<T>::operator - (T val) const {
  device_matrix<T> m(*this);
  return (m -= val);
}

template <typename T>
device_matrix<T>& device_matrix<T>::operator -= (const device_matrix<T>& rhs) {
  *this = *this - rhs;
  return *this;
}

template <typename T>
device_matrix<T> device_matrix<T>::operator - (const device_matrix<T>& rhs) const {
  device_matrix<T> result(_rows, _cols);
  sgeam(*this, rhs, result, 1.0, -1.0);
  return result;
}

// ===== Division =====
template <typename T>
device_matrix<T>& device_matrix<T>::operator /= (T alpha) {
  return *this *= ( (T) 1 / alpha );
}

template <typename T>
device_matrix<T> device_matrix<T>::operator / (T alpha) const {
  return *this * ( (T) 1 / alpha );
}

// ===== Matrix-scalar Multiplication =====
template <typename T>
device_matrix<T>& device_matrix<T>::operator *= (T alpha) {
  hipblasStatus_t status;
  status = hipblasSscal(CUBLAS_HANDLE::getInstance(), _rows*_cols, &alpha, _data, STRIDE);
  CCE(status);
  return *this;
}

template <typename T>
device_matrix<T> device_matrix<T>::operator * (T alpha) const {
  device_matrix<T> result(*this);
  return result *= alpha;
}

// ===== Matrix-Matrix Multiplication =====
template <typename T>
device_matrix<T>& device_matrix<T>::operator *= (const device_matrix<T>& rhs) {
  *this = *this * rhs;
  return *this;
}

template <typename T>
device_matrix<T> device_matrix<T>::operator * (const device_matrix<T>& rhs) const {
  device_matrix<T> result(_rows, rhs._cols);
  sgemm(*this, rhs, result);
  return result;
}

// Operator Assignment:
// call copy constructor first, and swap with the temp variable
template <typename T>
device_matrix<T>& device_matrix<T>::operator = (device_matrix<T> rhs) {
  swap(*this, rhs);
  return *this;
}

template <typename T>
void device_matrix<T>::_init() {
  CCE(hipMalloc((void **)&_data, _rows * _cols * sizeof(T)));
}

template <typename T>
void device_matrix<T>::resize(size_t r, size_t c) {
  if (_rows == r && _cols == c)
    return;

  _rows = r;
  _cols = c;
  _init();
  fillwith(0);
}

template <typename T>
void device_matrix<T>::print(FILE* fid) const {

  T* data = new T[size()];
  CCE(hipMemcpy(data, _data, sizeof(T) * size(), hipMemcpyDeviceToHost));

  for (size_t i=0; i<_rows; ++i) {
    for (size_t j=0; j<_cols; ++j)
      fprintf(fid, "%.7f ", data[j*_rows + i]);
    fprintf(fid, "\n");
  }

  if (fid == stdout)
    fprintf(fid, "\n");

  delete [] data;
}

template <typename T>
void device_matrix<T>::fillwith(T val) {
  hipMemset(_data, 0, _rows * _cols * sizeof(T));
}

template <typename T>
void device_matrix<T>::save(const string& filename) const {
  FILE* fid = fopen(filename.c_str(), "w");
  if (fid == NULL)
    return;

  print(fid);
  fclose(fid);
}

template <>
void device_matrix<float>::cublas_gemm(
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k,
  float alpha,
  const float* A, int lda,
  const float* B, int ldb,
  float beta,
  float* C, int ldc) {
  CCE(hipblasSgemm(CUBLAS_HANDLE::getInstance(), transA, transB, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc));
}

template <>
void device_matrix<double>::cublas_gemm(
  hipblasOperation_t transA, hipblasOperation_t transB,
  int m, int n, int k,
  double alpha,
  const double* A, int lda,
  const double* B, int ldb,
  double beta,
  double* C, int ldc) {
  CCE(hipblasDgemm(CUBLAS_HANDLE::getInstance(), transA, transB, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc));
}

template <>
void device_matrix<float>::cublas_geam(
    hipblasOperation_t transA, hipblasOperation_t transB,
    int m, int n,
    float alpha, const float *A, int lda,
    float beta , const float *B, int ldb,
    float *C, int ldc) {
  CCE(hipblasSgeam(CUBLAS_HANDLE::getInstance(), transA, transB, m, n, &alpha, A, lda, &beta, B, ldb, C, ldc));
}

template <>
void device_matrix<double>::cublas_geam(
    hipblasOperation_t transA, hipblasOperation_t transB,
    int m, int n,
    double alpha, const double *A, int lda,
    double beta , const double *B, int ldb,
    double *C, int ldc) {
  CCE(hipblasDgeam(CUBLAS_HANDLE::getInstance(), transA, transB, m, n, &alpha, A, lda, &beta, B, ldb, C, ldc));
}

template <>
void device_matrix<float>::cublas_nrm2(int n, const float *x, int incx, float *result) {
  CCE(hipblasSnrm2(CUBLAS_HANDLE::getInstance(), n, x, 1, result));
}

template <>
void device_matrix<double>::cublas_nrm2(int n, const double *x, int incx, double *result) {
  CCE(hipblasDnrm2(CUBLAS_HANDLE::getInstance(), n, x, 2, result));
}

// ++++++++++++++++++++++++++++++++++++++++++++
// +++++ Template Explicit Initialization +++++
// ++++++++++++++++++++++++++++++++++++++++++++
template class device_matrix<float>;

#define _DSMAT_ device_matrix<float>
float snrm2(const _DSMAT_& A) {
  float result;
  hipblasStatus_t status;
  status = hipblasSnrm2(CUBLAS_HANDLE::getInstance(), A.size(), A.getData(), 1, &result);
  CCE(status);
  return result;
}

void sgemm(const _DSMAT_& A, const _DSMAT_& B, _DSMAT_& C, float alpha, float beta) {
  // Perform C = αA*B + βC, not transpose on A and B
  size_t m = A.getRows();
  size_t n = B.getCols();
  C.resize(m, n);

  size_t k = A.getCols();

  int lda = A.getRows();
  int ldb = B.getRows();
  int ldc = C.getRows();

  hipblasStatus_t status;
  status = hipblasSgemm(CUBLAS_HANDLE::getInstance(), HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A.getData(), lda, B.getData(), ldb, &beta, C.getData(), ldc);

  CCE(status);
}

void sgeam(const _DSMAT_& A, const _DSMAT_& B, _DSMAT_& C, float alpha, float beta) {
  // Perform C = αA + βB, not transpose on A and B
  assert(A.getRows() == B.getRows() && A.getCols() == B.getCols());
  
  size_t m = A.getRows();
  size_t n = A.getCols();
  C.resize(m, n);

  int lda = A.getRows();
  int ldb = B.getRows();
  int ldc = C.getRows();

  hipblasStatus_t status;
  status = hipblasSgeam(CUBLAS_HANDLE::getInstance(), HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &alpha, A.getData(), lda, &beta, B.getData(), ldb, C.getData(), ldc);
  CCE(status);
}
#undef _DSMAT_

