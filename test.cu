#include "hip/hip_runtime.h"
#include <iostream>
#include <matrix.h>

#include <device_matrix.h>
#include <device_math_ext.h>
#include <device_arithmetic.h>
#define TEST_CL2E(x, y) {printf("checking "#x" ... "); compareL2error((x), (y));}

using namespace std;

typedef device_matrix<float> mat;
typedef thrust::device_vector<float> vec;

void testing();
void checkErrorIsAcceptable(double err);
void compareL2error(const vec& v, const vec& ref);
void compareL2error(const mat& m, const mat& ref);

int main (int argc, char* argv[]) {
  testing();
  return 0;
}

void testing() {
  mat A("data/A.mat");
  mat B("data/B.mat");
  mat C("data/C.mat");
  mat D("data/D.mat");

  vec x = ext::load<float>("data/x.vec");
  vec y = ext::load<float>("data/y.vec");
  vec u = ext::load<float>("data/u.vec");
  vec v = ext::load<float>("data/v.vec");

  mat ApB("data/A+B.mat");
  mat AmB("data/A-B.mat");
  mat CmD("data/C-D.mat");
  mat CpD("data/C+D.mat");

  mat AC("data/AC.mat");
  mat AD("data/AD.mat");
  mat BC("data/BC.mat");
  mat BD("data/BD.mat");

  vec Ax = ext::load<float>("data/Ax.vec");
  vec Bx = ext::load<float>("data/Bx.vec");
  vec Cy = ext::load<float>("data/Cy.vec");
  vec Dy = ext::load<float>("data/Dy.vec");

  vec uA = ext::load<float>("data/uA.vec");
  vec uB = ext::load<float>("data/uB.vec");
  vec vC = ext::load<float>("data/vC.vec");
  vec vD = ext::load<float>("data/vD.vec");

  mat PIpA("data/pi+A.mat");
  mat PImB("data/pi-B.mat");
  mat eC("data/eC.mat");
  mat D_over_e("data/D_over_e.mat");

  mat uAx("data/uAx.scalar");
  mat uBx("data/uBx.scalar");
  mat vCy("data/vCy.scalar");
  mat vDy("data/vDy.scalar");

  mat xu("data/xu.mat");
  mat xv("data/xv.mat");
  mat yu("data/yu.mat");
  mat yv("data/yv.mat");

  printf("A : %lu by %lu \n", A.getRows(), A.getCols());
  printf("A : %lu by %lu \n", C.getRows(), C.getCols());
  printf("AC: %lu by %lu \n", AC.getRows(), AC.getCols());

  printf("\n===== FILE I/O Testing =====\n");
  A.save("/tmp/cumatrix.mat");
  TEST_CL2E(mat("/tmp/cumatrix.mat"), A)

  printf("\n===== Matrix Addition =====\n");
  TEST_CL2E(A + B, ApB);
  TEST_CL2E(A - B, AmB);
  TEST_CL2E(C + D, CpD);
  TEST_CL2E(C - D, CmD);

  printf("\n===== Matrix - Matrix Multiplication =====\n");
  TEST_CL2E(A * C, AC);
  TEST_CL2E(A * D, AD);
  TEST_CL2E(B * C, BC);
  TEST_CL2E(B * D, BD);

  printf("\n===== Matrix - Vector Multiplication =====\n");
  TEST_CL2E(A * x, Ax);
  TEST_CL2E(B * x, Bx);
  TEST_CL2E(C * y, Cy);
  TEST_CL2E(D * y, Dy);

  TEST_CL2E(u * A, uA);
  TEST_CL2E(u * B, uB);
  TEST_CL2E(v * C, vC);
  TEST_CL2E(v * D, vD);

  printf("\n===== Matrix - Scalar Arithmetic =====\n");
  TEST_CL2E(A + PI, PIpA);
  // TEST_CL2E(PI - B, PImB);
  TEST_CL2E(2.718281828f * C, eC);
  TEST_CL2E(D / 2.718281828f, D_over_e);
}

void compareL2error(const mat& m, const mat& ref) {
  float error = snrm2(m - ref) / snrm2(ref);
  checkErrorIsAcceptable(error);
}

void compareL2error(const vec& v, const vec& ref) {
  float error = norm(v - ref) / norm(ref);
  checkErrorIsAcceptable(error);
}

void checkErrorIsAcceptable(double error) {
  const float EPS = 1e-6;
  if (error < EPS)
    printf("\33[32m[ OK ]\33[0m \n");
  else
    printf("error = %.4e > EPS (%.4e) \33[31m[FAILED]\33[0m\n", error, EPS);
}
